#include "hip/hip_runtime.h"
#include "device_funcs.cuh"
#include <hip/hip_vector_types.h>
#define EPS 0.01f

__host__ int divUp(int a, int b) { return (a + b - 1)/b; }

__device__
unsigned char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n); }

__device__ int clipWithBounds(int n, int n_min, int n_max) {
  return n > n_max ? n_max : (n < n_min ? n_min : n);
}

__device__ float3 yRotate(float3 pos, float theta) {
  const float c = cosf(theta), s = sinf(theta);
  return make_float3(c*pos.x + s*pos.z, pos.y, -s*pos.x + c*pos.z);
}

__device__ float3 scrIdxToPos(int c, int r, int w, int h, float zs) {
  return make_float3(c - w / 2, r - h / 2, zs);
}

__device__ float3 paramRay(Ray r, float t) { return r.o + t*(r.d); }

__device__ float planeSDF(float3 pos, float3 norm, float d) {
  return dot(pos, normalize(norm)) - d;
}

__device__
bool rayPlaneIntersect(Ray myRay, float3 n, float dist, float *t) {
  const float f0 = planeSDF(paramRay(myRay, 0.f), n, dist);
  const float f1 = planeSDF(paramRay(myRay, 1.f), n, dist);
  bool result = (f0*f1 < 0);
  if (result) *t = (0.f - f0) / (f1 - f0);
  return result;
}

// Intersect ray with a box from volumeRender SDK sample.
__device__ bool intersectBox(Ray r, float3 boxmin, float3 boxmax,
  float *tnear, float *tfar) {
  // Compute intersection of ray with all six bbox planes.
  const float3 invR = make_float3(1.0f) / r.d;
  const float3 tbot = invR*(boxmin - r.o), ttop = invR*(boxmax - r.o);
  // Re-order intersections to find smallest and largest on each axis.
  const float3 tmin = fminf(ttop, tbot), tmax = fmaxf(ttop, tbot);
  // Find the largest tmin and the smallest tmax.
  *tnear = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
  *tfar = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));
  return *tfar > *tnear;
}

__device__ int3 posToVolIndex(float3 pos, int3 volSize) {
  return make_int3(pos.x + volSize.x/2, pos.y + volSize.y/2,
                   pos.z + volSize.z/2);
}

__device__ int flatten(int3 index, int3 volSize) {
  return index.x + index.y*volSize.x + index.z*volSize.x*volSize.y;
}

__device__ float density(float2 *d_vol, int3 volSize, float3 pos, int method, int id) {
  int3 index = posToVolIndex(pos, volSize);
  int i = index.x, j = index.y, k = index.z;
  //printf("%d,%d,%d")
  const int w = volSize.x, h = volSize.y, d = volSize.z;
  const float3 rem = fracf(pos);
  index = make_int3(clipWithBounds(i, 0, w - 2),
    clipWithBounds(j, 0, h - 2), clipWithBounds(k, 0, d - 2));
  // directed increments for computing the gradient
  const int3 dx = { 1, 0, 0 }, dy = { 0, 1, 0 }, dz = { 0, 0, 1 };
  // values sampled at surrounding grid points
  if ((method == 1) && (id == 2)){
    const float dens000 = d_vol[flatten(index, volSize)].y;
    const float dens100 = d_vol[flatten(index + dx, volSize)].y;
    const float dens010 = d_vol[flatten(index + dy, volSize)].y;
    const float dens001 = d_vol[flatten(index + dz, volSize)].y;
    const float dens110 = d_vol[flatten(index + dx + dy, volSize)].y;
    const float dens101 = d_vol[flatten(index + dx + dz, volSize)].y;
    const float dens011 = d_vol[flatten(index + dy + dz, volSize)].y;
    const float dens111 = d_vol[flatten(index + dx + dy + dz, volSize)].y;
    // trilinear interpolation
    return (1 - rem.x)*(1 - rem.y)*(1 - rem.z)*dens000 +
      (rem.x)*(1 - rem.y)*(1 - rem.z)*dens100 +
      (1 - rem.x)*(rem.y)*(1 - rem.z)*dens010 +
      (1 - rem.x)*(1 - rem.y)*(rem.z)*dens001 +
      (rem.x)*(rem.y)*(1 - rem.z)*dens110 +
      (rem.x)*(1 - rem.y)*(rem.z)*dens101 +
      (1 - rem.x)*(rem.y)*(rem.z)*dens011 +
      (rem.x)*(rem.y)*(rem.z)*dens111;
  } else {
    const float dens000 = d_vol[flatten(index, volSize)].x;
    const float dens100 = d_vol[flatten(index + dx, volSize)].x;
    const float dens010 = d_vol[flatten(index + dy, volSize)].x;
    const float dens001 = d_vol[flatten(index + dz, volSize)].x;
    const float dens110 = d_vol[flatten(index + dx + dy, volSize)].x;
    const float dens101 = d_vol[flatten(index + dx + dz, volSize)].x;
    const float dens011 = d_vol[flatten(index + dy + dz, volSize)].x;
    const float dens111 = d_vol[flatten(index + dx + dy + dz, volSize)].x;
    // trilinear interpolation
    return (1 - rem.x)*(1 - rem.y)*(1 - rem.z)*dens000 +
      (rem.x)*(1 - rem.y)*(1 - rem.z)*dens100 +
      (1 - rem.x)*(rem.y)*(1 - rem.z)*dens010 +
      (1 - rem.x)*(1 - rem.y)*(rem.z)*dens001 +
      (rem.x)*(rem.y)*(1 - rem.z)*dens110 +
      (rem.x)*(1 - rem.y)*(rem.z)*dens101 +
      (1 - rem.x)*(rem.y)*(rem.z)*dens011 +
      (rem.x)*(rem.y)*(rem.z)*dens111;
  } 
}

__device__ uchar4 sliceShader(float2 *d_vol, int3 volSize, Ray boxRay,
  float gain, float dist, float3 norm, int id) {
  float t;
  uchar4 shade = make_uchar4(220, 220, 220, 0); // background value
  if (rayPlaneIntersect(boxRay, norm, dist, &t)) {
    float sliceDens = density(d_vol, volSize, paramRay(boxRay, t), 1, id);
    shade = make_uchar4(220, 220, 220, 0);
    if (id == 2){
      if (sliceDens>=0.1){shade = make_uchar4(clip(100+300*sliceDens), clip(10/(sliceDens)), clip(10/(sliceDens)), 0);}
      else if (sliceDens<=(-0.3)){shade = make_uchar4(clip(10/(-sliceDens)), clip(10/(-sliceDens)), clip(100+300*(-sliceDens)), 0);}
      else if (sliceDens>=(-0.3)){shade = make_uchar4(220, 220, 220, 0);}
    } else {
      shade = make_uchar4(101, 0, 222, 255);
      if (sliceDens>=0.7){shade = make_uchar4(200, 200, 200, 0);}
    }
  } 
    return shade;
}

__device__ uchar4 rayCastShader(float2 *d_vol, int3 volSize,
  Ray boxRay, float dist) {
  uchar4 shade = make_uchar4(200, 200, 200, 0);
  float3 pos = boxRay.o;
  float len = length(boxRay.d);
  float t = 0.0f;
  float f = density(d_vol, volSize, pos, 2, 1);
  while (f > dist + EPS && t < 1.0f) {
    f = density(d_vol, volSize, pos, 2, 1);
    t += (f - dist) / len;
    pos = paramRay(boxRay, t);
    f = density(d_vol, volSize, pos, 2, 1);
  }
  if (t < 1.f) {
    const float3 ux = make_float3(1, 0, 0), uy = make_float3(0, 1, 0),
                 uz = make_float3(0, 0, 1);
    float3 grad = {(density(d_vol, volSize, pos + EPS*ux, 2, 1) -
                    density(d_vol, volSize, pos, 2, 1))/EPS,
                   (density(d_vol, volSize, pos + EPS*uy, 2, 1) -
                   density(d_vol, volSize, pos, 2, 1))/EPS,
                   (density(d_vol, volSize, pos + EPS*uz, 2, 1) -
                   density(d_vol, volSize, pos, 2, 1))/EPS};
    float intensity = -dot(normalize(boxRay.d), normalize(grad));
    shade = make_uchar4(125 * intensity, 0 * intensity, 255* intensity, 255);
  }
  return shade;
}
